#include <iostream>
#include <hip/hip_runtime.h>
#include <memory>
#include <vector>

template <typename T>
class Cc_dev_ptr {
    static void freecuda(void* ptr) { hipFree(ptr); }
    std::unique_ptr<T[], decltype(&freecuda)> elem;
public:
    T* get() const { return elem.get(); }
    Cc_dev_ptr(const std::size_t N) : elem{nullptr, freecuda} {
        T* tmp;
        hipMalloc(reinterpret_cast<void**>(&tmp), N * sizeof(T));
        elem.reset(tmp);
    }
};

__global__ void fillIndices(int* data, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data[idx] = idx;
    }
}

int main() {
    const int N = 10;

    Cc_dev_ptr<int> device_data(N);
    std::vector<int> host_data(N);

    fillIndices<<<(N + 255) / 256, 256>>>(device_data.get(), N);
    hipMemcpy(host_data.data(), device_data.get(), N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        std::cout << "host_data[" << i << "] = " << host_data[i] << std::endl;
    }

    // No need to call cudaFree; Cc_dev_ptr will automatically release the memory.

    return 0;
}
 
